#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>


#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "../inc/utils.h"

/** 
 * Controle des erreurs CUDA et debugging. 
 */
#ifdef CUDA_DEBUG
#define CUDA_SYNC_ERROR() {						\
    hipError_t sync_error;						\
    hipDeviceSynchronize();						\
    sync_error = hipGetLastError();					\
    if(sync_error != hipSuccess) {					\
      fprintf(stderr, "[CUDA SYNC ERROR at %s:%d -> %s]\n",		\
	      __FILE__ , __LINE__, hipGetErrorString(sync_error));	\
      exit(EXIT_FAILURE);						\
    }									\
  }
#else /* #ifdef CUDA_DEBUG */
#define CUDA_SYNC_ERROR()
#endif /* #ifdef CUDA_DEBUG */

#define CUDA_ERROR(cuda_call) {					\
    hipError_t error = cuda_call;				\
    if(error != hipSuccess){					\
      fprintf(stderr, "[CUDA ERROR at %s:%d -> %s]\n",		\
	      __FILE__ , __LINE__, hipGetErrorString(error));	\
      exit(EXIT_FAILURE);					\
    }								\
    CUDA_SYNC_ERROR();						\
  }


/**
 * Retourne le quotient entier superieur ou egal a "a/b".
 * D apres : CUDA SDK 4.1
 */

static int iDivUp(int a, int b){
  return ((a % b != 0) ? (a / b + 1) : (a / b));
}

/**
 *
 * Function naive_algo()
 *
 **/
__global__ void naive_algo(unsigned long **data, int n, int l, int h, unsigned long long *S_gpu){

  int a = blockDim.x * blockIdx.x + threadIdx.x;
  int b = blockDim.y * blockIdx.y + threadIdx.y;

   
  int c = 0, ymin = 0, aux = n/10;
  unsigned long long S_loc = 0;

  if ((a < n) && (b < n)){
  
  //On effectue le calcul uniquement pour a<b
  if (a < b){
    if(b == a+1)
      ymin = h;
    else{
      ymin = data[a+1][1];
      for(c = a+1; c < b; c++){
	if(data[c][1] < ymin)
	  ymin = data[c][1];
      } // c loop
    } // else loop
    S_loc = (data[b][0] - data[a][0]) * ymin; 

    // Maximum sur tous les threads de tous les blocs (tres lent en theorie)
    // Rajouter des maxs locaux !!
    atomicMax(S_gpu, S_loc);

      /*if (a%aux == 0)
      printf("%d %%...", (a*100/n)+10);*/
    
  } //a<b loop

  }//test bound loop
    return;
  }


  /**
 *
 * Function main
 *
 **/
int main(int argc, char **argv){

  printf("start\n");
  double debut=0.0, fin=0.0;
  unsigned long **data, **data_gpu;;
  unsigned long long S = 0, *S_gpu;;
  int res = 0, i= 0;
  int n = 0, l = 0, h = 0;
  int *n_gpu, *l_gpu, *h_gpu;
  
  if(argc != 2){
    printf("Usage: %s <path_of_data_file>\n", argv[0]);
    return -1;
  }
  char *name = argv[1];

  /* Read parameters */
  res = read_param(name, data, &n, &l, &h);
  if(res != 0){
    printf("read_param :\t ERROR\n");
    return -1;
  }
  
  /* Allocate data table */
  data = (unsigned long **) malloc(n * sizeof(unsigned long *));
  for(i = 0; i < n; i++)
    data[i] = (unsigned long *) malloc(2 * sizeof(unsigned long));
  
  /* Read coordinates from file */
  res = read_data(name, data, n);
  if(res != 0){
    printf("read_data :\t ERROR\n");
    return -1;
  }
  
  /*printf("\nn=%d l=%d h=%d\n", n, l, h);*/
  /*for(i = 0; i < n; i++){
    printf("%lu,%lu\n", data[i][0], data[i][1]);
    }*/

  printf("allocation GPU\n");
  
  /* Allocation GPU */
  hipMalloc((void **)&n_gpu, sizeof(int));
  hipMalloc((void **)&l_gpu, sizeof(int));
  hipMalloc((void **)&h_gpu, sizeof(int));
  hipMalloc((void **)&S_gpu, sizeof(unsigned long long));
  
  hipMalloc((void **)&data_gpu, n*sizeof(unsigned long *));
  for(i = 0; i < n; i++)
    hipMalloc((void **)&data_gpu[i], 2 * sizeof(unsigned long));

  printf("Transferts CPU -> GPU\n");
  
  /* Transferts CPU -> GPU (synchrones) */
  hipMemcpy(n_gpu, &n, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(l_gpu, &l, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(h_gpu, &h, sizeof(int), hipMemcpyHostToDevice);
  for(i=0; i<n; i++){
    hipMemcpy(&data_gpu[i][0], &data[i][0], sizeof(unsigned long), hipMemcpyHostToDevice);
    hipMemcpy(&data_gpu[i][1], &data[i][1], sizeof(unsigned long), hipMemcpyHostToDevice);
  }


  printf("Lancement kernel\n");
  /* Lancement de kernel */
  
  //On utilise n*n threads mais ils n'effectueront pas tous des calcls en raison de la contrainte i<j
  dim3 threadsParBloc(32, 32);
  dim3 tailleGrille(iDivUp(n,32), iDivUp(n, 32));
    
  
  /* Start timing */
  debut = my_gettimeofday();

  /* Do computation:  */

  naive_algo<<<tailleGrille, threadsParBloc>>>(data_gpu, n, l, h, S_gpu);

  /* Recopie de l aire maximale sur le CPU */
  hipMemcpy(&S, S_gpu, sizeof(unsigned long long), hipMemcpyDeviceToHost);

  /* Free */
  hipFree(l_gpu);
  hipFree(h_gpu);
  hipFree(n_gpu);
  hipFree(S_gpu);
  for(i=0; i<n; i++)
    hipFree(data_gpu[i]);
  hipFree(data_gpu);
  
  /* End timing */
  fin = my_gettimeofday();
  
  fprintf(stdout, "N = %d\t S = %llu\n", n, S);
  /*fprintf( stdout, "For n=%d: total computation time (with gettimeofday()) : %g s\n\n",
  n, fin - debut);*/
  fprintf( stdout, "%g\n",
	   fin - debut);
  
  return 0;
}
