#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>


#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "../inc/utils.h"

#define NUM_LOCALS 10000

/** 
 * Controle des erreurs CUDA et debugging. 
 */
#ifdef CUDA_DEBUG
#define CUDA_SYNC_ERROR() {						\
    hipError_t sync_error;						\
    hipDeviceSynchronize();						\
    sync_error = hipGetLastError();					\
    if(sync_error != hipSuccess) {					\
      fprintf(stderr, "[CUDA SYNC ERROR at %s:%d -> %s]\n",		\
	      __FILE__ , __LINE__, hipGetErrorString(sync_error));	\
      exit(EXIT_FAILURE);						\
    }									\
  }
#else /* #ifdef CUDA_DEBUG */
#define CUDA_SYNC_ERROR()
#endif /* #ifdef CUDA_DEBUG */

#define CUDA_ERROR(cuda_call) {					\
    hipError_t error = cuda_call;				\
    if(error != hipSuccess){					\
      fprintf(stderr, "[CUDA ERROR at %s:%d -> %s]\n",		\
	      __FILE__ , __LINE__, hipGetErrorString(error));	\
      exit(EXIT_FAILURE);					\
    }								\
    CUDA_SYNC_ERROR();						\
  }


/**
 * Retourne le quotient entier superieur ou egal a "a/b".
 * D apres : CUDA SDK 4.1
 */

static int iDivUp(int a, int b){
  return ((a % b != 0) ? (a / b + 1) : (a / b));
}
/**
 *
 * Function naive_algo()
 *
 **/
__global__ void naive_algo(unsigned long *abs, unsigned long *ord, int n, int l, int h, unsigned long long *S_gpu, unsigned long long *local_max, int num_locals){

  int a = blockDim.x * blockIdx.x + threadIdx.x;
  int b = blockDim.y * blockIdx.y + threadIdx.y;
  int li = (a * n + b)%num_locals;

  unsigned long long old_max;
  
  int c = 0, ymin = 0, aux = n/10;
  unsigned long long S_it; //= 0;
  
  if ((a < n) && (b < n)){
    
    //On effectue le calcul uniquement pour a<b
    if (a < b){
      
      if(b == a+1)
	ymin = h;
      
      else{
       
	ymin = ord[a+1];
	for(c = a+1; c < b; c++){
	  if(ord[c] < ymin)
	    ymin = ord[c];
	} // c loop
      } // else loop
      
      S_it = (abs[b] - abs[a]) * ymin;
      
      //Optimisation avec maximum locaux
      old_max = atomicMax(&local_max[li], S_it);
      
      if (old_max < S_it)
	atomicMax(S_gpu, S_it);
      
      //if (a%aux == 0)
      //printf("%d %%...", (a*100/n)+10);
    } //a<b loop

  } //test bound loop

  return;
}


  /**
 *
 * Function main
 *
 **/
int main(int argc, char **argv){
 
  double debut=0.0, fin=0.0;
  unsigned long **data, *abs_gpu, *ord_gpu;
  unsigned long long S = 0, *S_gpu, *local_max_gpu;
  
  int num_locals = NUM_LOCALS; // modulo = nombre de maximaux locaux
  int res = 0, i= 0;
  int n = 0, l = 0, h = 0;
  int *n_gpu, *l_gpu, *h_gpu;
  
  if(argc != 2){
    printf("Usage: %s <path_of_data_file>\n", argv[0]);
    return -1;
  }
  char *name = argv[1];

  /* Read parameters */
  res = read_param(name, data, &n, &l, &h);
  if(res != 0){
    printf("read_param :\t ERROR\n");
    return -1;
  }
  
  /* Allocate data table */
  data = (unsigned long **) malloc(2 * sizeof(unsigned long *));
  data[0] = (unsigned long *) malloc(n * sizeof(unsigned long));
  data[1] = (unsigned long *) malloc(n * sizeof(unsigned long));

  
  /* Read coordinates from file */
  res = read_data(name, data, n);
  if(res != 0){
    printf("read_data :\t ERROR\n");
    return -1;
  }
  
  /*printf("\nn=%d l=%d h=%d\n", n, l, h);*/
  /*for(i = 0; i < n; i++){
    printf("%lu,%lu\n", data[i][0], data[i][1]);
    }*/


  printf("Allocation GPU\n");

  /* Allocation GPU */
  hipMalloc((void **)&n_gpu, sizeof(int));
  hipMalloc((void **)&l_gpu, sizeof(int));
  hipMalloc((void **)&h_gpu, sizeof(int));
  hipMalloc((void **)&S_gpu, sizeof(unsigned long long));
  if(n_gpu == NULL || l_gpu == NULL || h_gpu == NULL || S_gpu == NULL)
    printf("Parameters allocation failed\n");
  

  hipMalloc((void **)&abs_gpu, n * sizeof(unsigned long));
  hipMalloc((void **)&ord_gpu, n * sizeof(unsigned long));

  hipMalloc((void **)&local_max_gpu, num_locals * sizeof(unsigned long long));
  
  printf("Transferts CPU -> GPU\n");
  
  /* Transferts CPU -> GPU (synchrones) */
  hipMemcpy(n_gpu, &n, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(l_gpu, &l, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(h_gpu, &h, sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(abs_gpu, data[0], n * sizeof(unsigned long), hipMemcpyHostToDevice);
  hipMemcpy(ord_gpu, data[1], n * sizeof(unsigned long), hipMemcpyHostToDevice);

  hipMemset(local_max_gpu, 0, num_locals * sizeof(unsigned long long));
  hipMemset(S_gpu, 0, sizeof(unsigned long long));
	
  printf("lancement kernel\n");
  
  /* Lancement de kernel */
  
  //On utilise n*n threads mais ils n'effectueront pas tous des calcls en raison de la contrainte i<j
  dim3 threadsParBloc(32, 32);
  dim3 tailleGrille(iDivUp(n,32), iDivUp(n, 32));
    
  
  /* Start timing */
  debut = my_gettimeofday();

  /* Do computation:  */
  
  printf("lancement\n");
  
  naive_algo<<<tailleGrille, threadsParBloc>>>(abs_gpu, ord_gpu, n, l, h, S_gpu, local_max_gpu, num_locals);

  printf("sortie kernel\n");

  hipDeviceSynchronize();
  
  /* Recopie de l aire maximale sur le CPU */
  hipMemcpy((void *)&S, S_gpu, sizeof(unsigned long long), hipMemcpyDeviceToHost);
  

  
  /* End timing */
  fin = my_gettimeofday();
  
  fprintf(stdout, "N = %d\t S = %llu\n", n, S);
  /*fprintf( stdout, "For n=%d: total computation time (with gettimeofday()) : %g s\n\n",
  n, fin - debut);*/
  fprintf( stdout, "%g\n",
	   fin - debut);

  printf("free\n");
  
  /* Free */
  free(data[0]);
  free(data[1]);
  free(data);
  
  hipFree(l_gpu);
  hipFree(h_gpu);
  hipFree(n_gpu);
  hipFree(S_gpu);

  hipFree(abs_gpu);
  hipFree(ord_gpu);
  hipFree(local_max_gpu);
  
  return 0;
}
