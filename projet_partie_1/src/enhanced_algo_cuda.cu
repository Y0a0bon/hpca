#include "hip/hip_runtime.h"
/*********************************
 * Fichier enhanced_algo_cuda.cu *
 *********************************/
#include <stdlib.h>
#include <stdio.h>

#include <sys/time.h>
#include "../inc/utils.h"
#include <hip/hip_runtime.h>


/** 
 * CUDA error control and debugging. 
 */
#ifdef CUDA_DEBUG
#define CUDA_SYNC_ERROR() {						\
    hipError_t sync_error;						\
    hipDeviceSynchronize();						\
    sync_error = hipGetLastError();					\
    if(sync_error != hipSuccess) {					\
      fprintf(stderr, "[CUDA SYNC ERROR at %s:%d -> %s]\n",		\
	      __FILE__ , __LINE__, hipGetErrorString(sync_error));	\
      exit(EXIT_FAILURE);						\
    }									\
  }
#else /* #ifdef CUDA_DEBUG */
#define CUDA_SYNC_ERROR()
#endif /* #ifdef CUDA_DEBUG */

#define CUDA_ERROR(cuda_call) {					\
    hipError_t error = cuda_call;				\
    if(error != hipSuccess){					\
      fprintf(stderr, "[CUDA ERROR at %s:%d -> %s]\n",		\
	      __FILE__ , __LINE__, hipGetErrorString(error));	\
      exit(EXIT_FAILURE);					\
    }								\
    CUDA_SYNC_ERROR();						\
  }


/**
 * Function iDivUp()
 * Return integer quotient superior or equal to "a/b"
 * Source : CUDA SDK 4.1
 */
static int iDivUp(int a, int b){
  return ((a % b != 0) ? (a / b + 1) : (a / b));
}


/**
 *
 * Function enhanced_algo_cuda()
 *
 **/
__global__ void enhanced_algo_cuda(unsigned long *abs, unsigned long *ord, int n, int l, int h, unsigned long long *S_gpu){

  
  int a = blockDim.x * blockIdx.x + threadIdx.x;
  int b;

  int ymin = 0, aux = n/10;
  unsigned long long S_loc = 0, S = 0;

 if ((a < n)){
    
   // Compute only when a < b
   for (b = a+1; b < n; b++){
      
      if(b == a+1)
	ymin = h;

      else if (ymin > ord[b-1])
	ymin = ord[b-1];
      // else -- nothing
      // WARNING : no default case

      S_loc = (abs[b] - abs[a]) * ymin;

      if(S_loc > S)
	S = S_loc;
      
   } // for loop
   
   // Maximum on every thread of every blocks (slow)
   atomicMax(S_gpu, S);
   
  } // test bound loop

  return;
}


/**
 *
 * Function main()
 *
 **/
int main(int argc, char **argv){
 
  double debut=0.0, fin=0.0;
  unsigned long **data, *abs_gpu, *ord_gpu;
  unsigned long long S = 0, *S_gpu;
  int res = 0, i= 0;
  int n = 0, l = 0, h = 0;
  int *n_gpu, *l_gpu, *h_gpu;
  
  if(argc != 2){
    printf("Usage: %s <path_of_data_file>\n", argv[0]);
    return -1;
  }
  char *name = argv[1];

  /* Read parameters */
  res = read_param(name, data, &n, &l, &h);
  if(res != 0){
    printf("read_param :\t ERROR\n");
    return -1;
  }
  
  /* Allocate data table */
  data = (unsigned long **) malloc(2 * sizeof(unsigned long *));
  data[0] = (unsigned long *) malloc(n * sizeof(unsigned long));
  data[1] = (unsigned long *) malloc(n * sizeof(unsigned long));
  
  /* Read coordinates from file */
  res = read_data(name, data, n);
  if(res != 0){
    printf("read_data :\t ERROR\n");
    return -1;
  }
  

  /* GPU allocation */
  printf("GPU allocation.\n");
  hipMalloc((void **)&n_gpu, sizeof(int));
  hipMalloc((void **)&l_gpu, sizeof(int));
  hipMalloc((void **)&h_gpu, sizeof(int));
  hipMalloc((void **)&S_gpu, sizeof(unsigned long long));
  if(n_gpu == NULL || l_gpu == NULL || h_gpu == NULL || S_gpu == NULL)
    printf("Parameters allocation failed\n");


  hipMalloc((void **)&abs_gpu, n * sizeof(unsigned long));
  hipMalloc((void **)&ord_gpu, n * sizeof(unsigned long));

  
  /* CPU -> GPU transfer (synchrones) */
  printf("CPU -> GPU transfer.\n");
  hipMemcpy(n_gpu, &n, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(l_gpu, &l, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(h_gpu, &h, sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(abs_gpu, data[0], n * sizeof(unsigned long), hipMemcpyHostToDevice);
  hipMemcpy(ord_gpu, data[1], n * sizeof(unsigned long), hipMemcpyHostToDevice);

  hipMemset(S_gpu, 0, sizeof(unsigned long long));
  

  /* Kernel launching */
  printf("Launching kernel.\n");
  // Using n*n threads but not every one is useful, because of the "i < j" constraint
  dim3 threadsParBloc(32, 1);
  dim3 tailleGrille(iDivUp(n,32), 1);
  
  
  /* Start timing */
  debut = my_gettimeofday();
  
  /* Do computation:  */

  printf("Lauching.\n");  
  enhanced_algo_cuda<<<tailleGrille, threadsParBloc>>>(abs_gpu, ord_gpu, n, l, h, S_gpu);

  printf("Leaving kernel.\n");
  
  hipDeviceSynchronize();

  /* Recovering maximal area on CPU */
  hipMemcpy((void *)&S, S_gpu, sizeof(unsigned long long), hipMemcpyDeviceToHost);
  
  
  /* End timing */
  fin = my_gettimeofday();

  fprintf(stdout, "***** Algorithme amélioré, en CUDA *****\n");
  fprintf(stdout, "Pour les paramètres N = %d\t S = %llu\n", n, S);
  fprintf( stdout, "Total computation time in s (with gettimeofday()) :\t");
  fprintf( stdout, "%g\n\n",
	   fin - debut);

  
   /* Free */
  printf("\nFreeing and quitting.\n");
  free(data[0]);
  free(data[1]);
  free(data);
  
  hipFree(l_gpu);
  hipFree(h_gpu);
  hipFree(n_gpu);
  hipFree(S_gpu);

  hipFree(abs_gpu);
  hipFree(ord_gpu);

  return 0;
}
