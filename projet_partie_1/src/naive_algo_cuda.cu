#include "hip/hip_runtime.h"
/******************************
 * Fichier naive_algo_cuda.cu *
 ******************************/
#include <stdlib.h>
#include <stdio.h>

#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "../inc/utils.h"


/** 
 * CUDA error control and debugging. 
 */
#ifdef CUDA_DEBUG
#define CUDA_SYNC_ERROR() {						\
    hipError_t sync_error;						\
    hipDeviceSynchronize();						\
    sync_error = hipGetLastError();					\
    if(sync_error != hipSuccess) {					\
      fprintf(stderr, "[CUDA SYNC ERROR at %s:%d -> %s]\n",		\
	      __FILE__ , __LINE__, hipGetErrorString(sync_error));	\
      exit(EXIT_FAILURE);						\
    }									\
  }
#else /* #ifdef CUDA_DEBUG */
#define CUDA_SYNC_ERROR()
#endif /* #ifdef CUDA_DEBUG */

#define CUDA_ERROR(cuda_call) {					\
    hipError_t error = cuda_call;				\
    if(error != hipSuccess){					\
      fprintf(stderr, "[CUDA ERROR at %s:%d -> %s]\n",		\
	      __FILE__ , __LINE__, hipGetErrorString(error));	\
      exit(EXIT_FAILURE);					\
    }								\
    CUDA_SYNC_ERROR();						\
  }


/**
 * Function iDivUp()
 * Return integer quotient superior or equal to "a/b"
 * Source : CUDA SDK 4.1
 */
static int iDivUp(int a, int b){
  return ((a % b != 0) ? (a / b + 1) : (a / b));
}


/**
 *
 * Function naive_algo_cuda()
 *
 **/
__global__ void naive_algo_cuda(unsigned long *abs, unsigned long *ord, int n, int l, int h, unsigned long long *S_gpu){

  int a = blockDim.x * blockIdx.x + threadIdx.x;
  int b = blockDim.y * blockIdx.y + threadIdx.y;

  int c = 0, ymin = 0, aux = n/10;
  unsigned long long S_loc; //= 0;

  
  if ((a < n) && (b < n)){
    
    // Compute only when a < b
    if (a < b){
      
      if(b == a+1)
	ymin = h;
      
      else{
       
	ymin = ord[a+1];
	for(c = a+1; c < b; c++){
	  if(ord[c] < ymin)
	    ymin = ord[c];
	} // c loop
      } // else loop
      
      S_loc = (abs[b] - abs[a]) * ymin;
      
      // Maximum on every thread of every blocks (slow)
      atomicMax(S_gpu, S_loc);
      
    } //a<b loop

  } //test bound loop

  return;
}


  /**
 *
 * Function main()
 *
 **/
int main(int argc, char **argv){
 
  double debut=0.0, fin=0.0;
  unsigned long **data, *abs_gpu, *ord_gpu;
  unsigned long long S = 0, *S_gpu;
  int res = 0, i= 0;
  int n = 0, l = 0, h = 0;
  int *n_gpu, *l_gpu, *h_gpu;
  
  if(argc != 2){
    printf("Usage: %s <path_of_data_file>\n", argv[0]);
    return -1;
  }
  char *name = argv[1];

  /* Read parameters */
  res = read_param(name, data, &n, &l, &h);
  if(res != 0){
    printf("read_param :\t ERROR\n");
    return -1;
  }
  
  /* Allocate data table */
  data = (unsigned long **) malloc(2 * sizeof(unsigned long *));
  data[0] = (unsigned long *) malloc(n * sizeof(unsigned long));
  data[1] = (unsigned long *) malloc(n * sizeof(unsigned long));

  
  /* Read coordinates from file */
  res = read_data(name, data, n);
  if(res != 0){
    printf("read_data :\t ERROR\n");
    return -1;
  }
  
  /*printf("\nn=%d l=%d h=%d\n", n, l, h);*/
  /*for(i = 0; i < n; i++){
    printf("%lu,%lu\n", data[i][0], data[i][1]);
    }*/


  /* GPU allocation */
  printf("GPU allocation.\n");
  hipMalloc((void **)&n_gpu, sizeof(int));
  hipMalloc((void **)&l_gpu, sizeof(int));
  hipMalloc((void **)&h_gpu, sizeof(int));
  hipMalloc((void **)&S_gpu, sizeof(unsigned long long));
  if(n_gpu == NULL || l_gpu == NULL || h_gpu == NULL || S_gpu == NULL)
    printf("Parameters allocation failed\n");


  hipMalloc((void **)&abs_gpu, n * sizeof(unsigned long));
  hipMalloc((void **)&ord_gpu, n * sizeof(unsigned long));

  /* CPU -> GPU transfer (synchrones) */
  printf("CPU -> GPU transfer.\n");
  hipMemcpy(n_gpu, &n, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(l_gpu, &l, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(h_gpu, &h, sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(abs_gpu, data[0], n * sizeof(unsigned long), hipMemcpyHostToDevice);
  hipMemcpy(ord_gpu, data[1], n * sizeof(unsigned long), hipMemcpyHostToDevice);

  hipMemset(S_gpu, 0, sizeof(unsigned long long));

  
  /* Kernel launching */
  printf("Launching kernel.\n");
  // Using n*n threads but not every one is useful, because of the "i < j" constraint
  dim3 threadsParBloc(32, 32);
  dim3 tailleGrille(iDivUp(n,32), iDivUp(n, 32));

  
  /* Start timing */
  debut = my_gettimeofday();

  /* Do computation:  */
  
  printf("Lauching.\n");
  naive_algo_cuda<<<tailleGrille, threadsParBloc>>>(abs_gpu, ord_gpu, n, l, h, S_gpu);

  printf("Leaving kernel.\n");

  hipDeviceSynchronize();
  
  /* Recovering maximal area on CPU */
  hipMemcpy((void *)&S, S_gpu, sizeof(unsigned long long), hipMemcpyDeviceToHost);
  

  /* End timing */
  fin = my_gettimeofday();
  
  fprintf(stdout, "N = %d\t S = %llu\n", n, S);
  fprintf( stdout, "For n=%d: total computation time in s (with gettimeofday()) :\n",
  n);
  fprintf( stdout, "%g\n",
	   fin - debut);

  
  /* Free */
  printf("\nFreeing and quitting.\n");
  free(data[0]);
  free(data[1]);
  free(data);
  
  hipFree(l_gpu);
  hipFree(h_gpu);
  hipFree(n_gpu);
  hipFree(S_gpu);

  hipFree(abs_gpu);
  hipFree(ord_gpu);
  
  return 0;
}
