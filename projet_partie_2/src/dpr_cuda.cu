#include "hip/hip_runtime.h"
/*********************************
 * Fichier dpr_cuda.cu *
 *********************************/
#include <stdlib.h>
#include <stdio.h>

#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "../inc/utils.h"

//Min nb of points to launch the GPU computation
#define TRESHOLD_SEQ 90000

//Nb points in each parallel region
#define SIZE_PARALLEL 50000

/**
 * CUDA error control and debugging.
 **/
#ifdef CUDA_DEBUG
#define CUDA_SYNC_ERROR() {						\
    hipError_t sync_error;						\
    hipDeviceSynchronize();						\
    Sync_error = hipGetLastError();					\
    if(sync_error != hipSuccess) {					\
      fprintf(stderr, "[CUDA SYNC ERROR at %s:%d -> %s]\n",		\
	      __FILE__ , __LINE__, hipGetErrorString(sync_error));	\
      exit(EXIT_FAILURE);						\
    }									\
  }
#else /* #ifdef CUDA_DEBUG */
#define CUDA_SYNC_ERROR()
#endif /* #ifdef CUDA_DEBUG */

#define CUDA_ERROR(cuda_call) {					\
    hipError_t error = cuda_call;				\
    if(error != hipSuccess){					\
      fprintf(stderr, "[CUDA ERROR at %s:%d -> %s]\n",		\
	      __FILE__ , __LINE__, hipGetErrorString(error));	\
      exit(EXIT_FAILURE);					\
    }								\
    CUDA_SYNC_ERROR();						\
  }

__global__ void calcul_min( unsigned long *ord, int ind_start, int ind_end, unsigned long long *ymin, int *ind_min, int size_max_parallel ){

  int a = threadIdx.x;
  int size_tot = (ind_end - ind_start -1);

  //On n'effectue pas le calcul aux indices ind_start ni ind_end
  int nb_threads = ceilf((float)size_tot/(float)size_max_parallel);

  //size of region to compute in the current thread
  int size_parallel = ceilf( (float)size_tot/(float)nb_threads );


  //have to be computed before the case of a different size_parallel value
  int ind_start_loc = ind_start + a * size_parallel + 1;
  
  if ( a == (nb_threads - 1) )
    size_parallel = size_tot - (nb_threads - 1) * size_parallel;


  unsigned long min_loc = ord[ind_start_loc];
  int ind_min_loc = ind_start_loc;
  int i = 0;

  //printf("FINDING YMIN\n");
  
  for ( i = ind_start_loc; i < ind_start_loc + size_parallel; i++ ){
    
    //Looking for the lowest ordinate
    if ( ord[i]< min_loc ){
      min_loc = ord[i];
      ind_min_loc = i;
      
    }

  }

  atomicMin(ymin, min_loc);
  
  __syncthreads();

  if (*ymin == min_loc)
    *ind_min = ind_min_loc;
  
  return;
}




/**
 *
 * Function dpr_cuda()
 *
 **/

unsigned long long dpr_cuda(unsigned long **data, int n, int l, unsigned long h, int ind_start, int ind_end){

  int i = 0;
  
  //ycross min on the whole area, ymin min on the whole area minus the 2 ends
  int ind_min = 0;
 
  unsigned long long crosswise_area = 0, left_area = 0, right_area = 0, result_area = 0, ymin =0;


  
  //Two points left : returns the rectangle defined by the height
  if ( (ind_end - ind_start) == 1 ){

    return (unsigned long long) (data[0][ind_end]-data[0][ind_start]) * h;
  }

  // No parallel computing if too few points
  if ( (ind_end - ind_start) < TRESHOLD_SEQ ){
    ymin = data[1][ind_start + 1];
    ind_min = ind_start + 1;
    
    
    //We don't enter the loop if ind_end - ind_start == 2
    for ( i = ind_start + 2; i < ind_end; i++ ){
      
      //Looking for the lowest ordinate
      if ( data[1][i] < ymin ){
	ymin = data[1][i];
	ind_min = i;
      }
    }
    
  }
  else {

    int *ind_min_gpu, *ind_start_gpu, *ind_end_gpu, size_parallel = SIZE_PARALLEL, *size_parallel_gpu;
    unsigned long *ord_gpu;
    unsigned long long *min_gpu;

    
    //INIT GPU PARAMETERS
    /* GPU allocation */
    hipMalloc((void **)&min_gpu, sizeof(unsigned long long));
    hipMalloc((void **)&ind_min_gpu, sizeof(int));
    hipMalloc((void **)&ind_start_gpu, sizeof(int));
    hipMalloc((void **)&ind_end_gpu, sizeof(int));
    hipMalloc((void **)&size_parallel_gpu, sizeof(int));
    if(min_gpu == NULL || ind_min_gpu == NULL || ind_start_gpu == NULL || ind_end_gpu == NULL || size_parallel_gpu == NULL)
      printf("Parameters allocation failed\n");
  
    hipMalloc((void **)&ord_gpu, n * sizeof(unsigned long));
  

    /* CPU -> GPU transfer (synchrones) */
    hipMemcpy(ord_gpu, data[1], n * sizeof(unsigned long), hipMemcpyHostToDevice);
    hipMemcpy(ind_start_gpu, &ind_start, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(ind_end_gpu, &ind_end, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(size_parallel_gpu, &size_parallel, sizeof(int), hipMemcpyHostToDevice);
  
    hipMemset(min_gpu, h, sizeof(unsigned long long));
    hipMemset(ind_min_gpu, -1, sizeof(int));
  
    /* Kernel launching */
    
    //Un seul bloc de threads 1D
    int size_tot = (ind_end - ind_start -1);
    int nb_threads = ceil((float)size_tot/(float)SIZE_PARALLEL);


    dim3 threadsParBloc(nb_threads, 1);
    dim3 tailleGrille(1, 1);
  
    // Compute ymin on GPU
    calcul_min<<<tailleGrille, threadsParBloc>>>(ord_gpu, ind_start, ind_end, min_gpu, ind_min_gpu, size_parallel);


    /* Recovering min element and index on CPU (element too for testing purposes) */
    hipMemcpy((void *)&ymin, min_gpu, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    hipMemcpy((void *)&ind_min, ind_min_gpu, sizeof(int), hipMemcpyDeviceToHost);


    /* cuda Frees */
    hipFree(min_gpu);
    hipFree(ind_min_gpu);
    hipFree(ind_start_gpu);
    hipFree(ind_end_gpu);
    hipFree(ord_gpu);

  }
  
  crosswise_area = ymin * (data[0][ind_end] - data[0][ind_start]);

  left_area = dpr_cuda(data, n, l, h, ind_start, ind_min);
  right_area = dpr_cuda(data, n, l, h, ind_min, ind_end);

  
  //Result is the max of these areas
  result_area = crosswise_area;
  if ( left_area > result_area )
    result_area = left_area;
  if ( right_area > result_area )
    result_area = right_area;

  
  return result_area;
  
}

int main(int argc, char **argv){

  double debut=0.0, fin=0.0;
  unsigned long **data;
  unsigned long long S = 0, h = 0;
  int res = 0;
  int n = 0, l = 0;

  if(argc != 2){
    printf("Usage: %s <path_of_data_file>\n", argv[0]);
    return -1;
  }
  char *name = argv[1];

  /* Read parameters */
  res = read_param_cuda(name, data, &n, &l, &h);
  if(res != 0){
    printf("read_param :\t ERROR\n");
    return -1;
  }
  
  /* Allocate data table */
  data = (unsigned long **) malloc(2 * sizeof(unsigned long *));
  data[0] = (unsigned long *) malloc(n * sizeof(unsigned long));
  data[1] = (unsigned long *) malloc(n * sizeof(unsigned long));
  
  /* Read coordinates from file */
  res = read_data(name, data, n);
  if(res != 0){
    printf("read_data :\t ERROR\n");
    return -1;
  }
  
  /* Start timing */
  debut = my_gettimeofday();

  /* Do computation:  */

  S = dpr_cuda(data, n, l, h, 0, n-1);
  
  /* End timing */
  fin = my_gettimeofday();
  fprintf(stdout, "\n***** Algorithme Diviser Pour Régner, hybride *****\n");
  fprintf(stdout, "Pour les paramètres N = %d\t S = %llu\nTRESHOLD_SEQ = %d\t, SIZE_PARALLEL = %d\n", n, S, TRESHOLD_SEQ, SIZE_PARALLEL);
  fprintf( stdout, "Total computation time in s (with gettimeofday()) :\t");
  fprintf( stdout, "%g\n\n",
	   fin - debut);

  return 0;
}
